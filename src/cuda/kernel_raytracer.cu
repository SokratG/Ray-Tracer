#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <types.hpp>
#include <random>
#include <cassert>
#include <stdint.h>

using byte = uint8_t;

struct RT_Data
{
	int64_t width = 0;
	int64_t height = 0;
	int64_t sample_per_pixel = 0;
	double gammacorrection = 2.0;
	int64_t maxdepth = 0;
	color backgroundcolor;
};

__global__ void cu_draw_kernel(byte* cu_framebuffer, const RT_Data& rt_data);

inline double cu_random_double() {
	static std::uniform_real_distribution<double> distribution(0.0, 1.0);
	static std::mt19937 generator;
	return distribution(generator);
}

void cu_render(byte* framebuffer, const size_t buffer_size, 
				const color backcolor, const int64_t width, const int64_t height,
				int64_t sample_per_pixel, int64_t maxdepth, const double gammacorrection)
{
	byte* cu_framebuffer = nullptr;
	hipError_t cudaStatus;
	
	RT_Data rt_data;
	rt_data.width = width;
	rt_data.height = height;
	rt_data.sample_per_pixel = sample_per_pixel;
	rt_data.gammacorrection = gammacorrection;
	rt_data.maxdepth = maxdepth;
	rt_data.backgroundcolor = backcolor;
	

	cudaStatus = hipSetDevice(0); // pick first GPU device
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error: set default GPU device: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}


	cudaStatus = hipMalloc(&cu_framebuffer, buffer_size * sizeof(byte));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error: memory allocation for GPU device: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}


	const uint32_t blockSize = rt_data.height;
	const uint32_t numBlocks = rt_data.width;
	
	// cu_render_kernel<<<dim3(numBlocks), dim3(blockSize)>>>(cu_framebuffer, world, rt_data);

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}


	cudaStatus = hipMemcpy(framebuffer, cu_framebuffer, buffer_size * sizeof(byte), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error: memory copy GPU-device to Host: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}


	cudaStatus = hipFree(cu_framebuffer);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Error: memory deallocation for GPU device: %s\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}

	
}

__global__ void cu_AA_RGBPixel(byte pixels[3], const vec3& lightcolor, const size_t sample_per_pixel, const double gamma)
{
	assert(gamma > 0);
	auto r = lightcolor[0];
	auto g = lightcolor[1];
	auto b = lightcolor[2];

	auto scale = 1.0 / sample_per_pixel;
	// Divide the color by the number of samples and gamma-correct for gamma correction
	auto gammacorrection = 1.0 / gamma;
	r = pow((r * scale), gammacorrection);
	g = pow((g * scale), gammacorrection);
	b = pow((b * scale), gammacorrection);

	pixels[0] = static_cast<byte>(__saturatef(r) * 255);
	pixels[1] = static_cast<byte>(__saturatef(g) * 255);
	pixels[2] = static_cast<byte>(__saturatef(b) * 255);
}


